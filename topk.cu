#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) <year> NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "topk.h"

#define CHECK(call)                                                          \
    {                                                                        \
        const hipError_t error = call;                                      \
        if (error != hipSuccess) {                                          \
            printf("ERROR: %s:%d,", __FILE__, __LINE__);                     \
            printf("code:%d,reason:%s\n", error, hipGetErrorString(error)); \
            exit(1);                                                         \
        }                                                                    \
    }

int show_mem_usage() {
    hipError_t err;
    // show memory usage of GPU
    size_t free_byte;
    size_t total_byte;
    err = hipMemGetInfo(&free_byte, &total_byte);
    CUDA_CHECK(err, "check memory info.");
    size_t used_byte = total_byte - free_byte;
    printf("GPU memory usage: used = %4.2lf MB, free = %4.2lf MB, total = %4.2lf MB\n",
           used_byte / 1024.0 / 1024.0, free_byte / 1024.0 / 1024.0, total_byte / 1024.0 / 1024.0);
    return hipSuccess;
}

int getThreadNum() {
    hipDeviceProp_t prop;
    int count;

    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n", count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
    printf("max thread num: %d\n", prop.maxThreadsPerBlock);
    printf("max grid dimensions: %d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

typedef uint4 group_t;  // cuda uint4: 4 * uint (64bit, sizeof(uint4)=16 256bit)

// intersection(query,doc): query[i] == doc[j](0 <= i < query_size, 0 <= j < doc_size)
// score = total_intersection(query,doc) / max(query_size, doc_size)
// note: query/doc vec must sorted by ASC
void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs,
    const int *doc_lens, const size_t n_docs,
    uint16_t *query, const int query_len, float *scores) {
#ifdef DEBUG
    printf("tid:%d GPU from block(%d, %d, %d), thread(%d, %d, %d)\n ",
           tid,
           blockIdx.x,
           blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z);
#endif
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;
    if (tid >= n_docs) {
        return;
    }

    __shared__ uint16_t query_on_shm[MAX_QUERY_SIZE];
#pragma unroll
    for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
        query_on_shm[i] = query[i];  // not very efficient query loading temporally, as assuming its not hotspot
    }

    __syncthreads();

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register int query_idx = 0;
        register float tmp_score = 0.;
        register bool no_more_load = false;

        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) {
                break;
            }
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
            register uint16_t *doc_segment = (uint16_t *)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                    // return;
                }
                // todo: hashmap/bitmap (just for int, but if embedding float/double don't ok)
                while (query_idx < query_len && query_on_shm[query_idx] < doc_segment[j]) {
                    ++query_idx;
                }
                if (query_idx < query_len) {
                    tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
                }
            }
            __syncwarp();
        }
        scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    }
}

void doc_query_scoring_gpu(std::vector<std::vector<uint16_t>> &querys,
                           int start_doc_id,
                           std::vector<std::vector<uint16_t>> &docs,
                           std::vector<uint16_t> &lens,
                           std::vector<std::vector<int>> &indices,  // shape [querys.size(), TOPK]
                           std::vector<std::vector<float>> &scores  // shape [querys.size(), TOPK]
) {
    auto n_docs = docs.size();
    std::vector<float> s_scores(n_docs);
    std::vector<int> s_indices(n_docs);

    float *d_scores = nullptr;
    uint16_t *d_docs = nullptr, *d_query = nullptr;
    int *d_doc_lens = nullptr;

    // copy to device
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    hipMalloc(&d_scores, sizeof(float) * n_docs);
    hipMalloc(&d_doc_lens, sizeof(int) * n_docs);

    // pre align docs -> h_docs [n_docs,MAX_DOC_SIZE], h_doc_lens_vec[n_docs]
    uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
    memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    std::vector<int> h_doc_lens_vec(n_docs);
    for (int i = 0; i < docs.size(); i++) {
        for (int j = 0; j < docs[i].size(); j++) {
            auto group_sz = sizeof(group_t) / sizeof(uint16_t);
            auto layer_0_offset = j / group_sz;
            auto layer_0_stride = n_docs * group_sz;
            auto layer_1_offset = i;
            auto layer_1_stride = group_sz;
            auto layer_2_offset = j % group_sz;
            auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
            h_docs[final_offset] = docs[i][j];
        }
        h_doc_lens_vec[i] = docs[i].size();
    }

    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    hipMemcpy(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs, hipMemcpyHostToDevice);

    // use one gpu device
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    hipSetDevice(0);

    for (auto &query : querys) {
        // init indices
        for (int i = 0; i < n_docs; ++i) {
            s_indices[i] = i + start_doc_id;
        }

        const size_t query_len = query.size();
        hipMalloc(&d_query, sizeof(uint16_t) * query_len);
        hipMemcpy(d_query, query.data(), sizeof(uint16_t) * query_len, hipMemcpyHostToDevice);
        show_mem_usage();

        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;

        std::chrono::high_resolution_clock::time_point tt = std::chrono::high_resolution_clock::now();
        // hipLaunchKernel
        docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block>>>(d_docs,
                                                                          d_doc_lens, n_docs, d_query, query_len, d_scores);
        hipDeviceSynchronize();
        hipMemcpy(s_scores.data(), d_scores, sizeof(float) * n_docs, hipMemcpyDeviceToHost);
        std::chrono::high_resolution_clock::time_point tt1 = std::chrono::high_resolution_clock::now();
        // std::cout << "docQueryScoringCoalescedMemoryAccessSampleKernel cost " << std::chrono::duration_cast<std::chrono::milliseconds>(tt1 - tt).count() << " ms " << std::endl;

        std::chrono::high_resolution_clock::time_point t = std::chrono::high_resolution_clock::now();
        int topk = s_scores.size() > TOPK ? TOPK : s_scores.size();
        // sort scores with Heap-based sort
        // todo: Bitonic sort by gpu
        std::partial_sort(s_indices.begin(), s_indices.begin() + topk, s_indices.end(),
                          [&s_scores, start_doc_id](const int &a, const int &b) {
                              if (s_scores[a - start_doc_id] != s_scores[b - start_doc_id]) {
                                  return s_scores[a - start_doc_id] > s_scores[b - start_doc_id];  // by score DESC
                              }
                              return a < b;  // the same score, by index ASC
                          });
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
        // std::cout << "heap partial_sort cost " << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t).count() << " ms " << std::endl;

        std::vector<int> topk_doc_ids(s_indices.begin(), s_indices.begin() + topk);
        indices.push_back(topk_doc_ids);

        std::vector<float> topk_scores(topk_doc_ids.size());
        int i = 0;
        for (auto doc_id : topk_doc_ids) {
            topk_scores[i++] = s_scores[doc_id - start_doc_id];
        }
        scores.push_back(topk_scores);

        hipFree(d_query);
    }

    // deallocation
    hipFree(d_docs);
    // hipFree(d_query);
    hipFree(d_scores);
    hipFree(d_doc_lens);
    free(h_docs);
}
