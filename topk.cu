#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) <year> NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "helper.h"
#include "topk.h"

// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#built-in-vector-types
typedef uint4 group_t;  // cuda uint4: 4 * uint (32it, sizeof(uint4)=16 128bit)

// intersection(query,doc): query[i] == doc[j](0 <= i < query_size, 0 <= j < doc_size)
// score = total_intersection(query,doc) / max(query_size, doc_size)
// note: query/doc vec must sorted by ASC
void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
    const __restrict__ uint16_t *docs,
    const int *doc_lens, const size_t n_docs,
    uint16_t *query, const int query_len, float *scores) {
#ifdef DEBUG
    printf("tid:%d GPU from block(%d, %d, %d), thread(%d, %d, %d)\n ",
           tid,
           blockIdx.x,
           blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z);
#endif
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;
    if (tid >= n_docs) {
        return;
    }

    __shared__ uint16_t query_on_shm[MAX_QUERY_SIZE];
#pragma unroll
    for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
        query_on_shm[i] = query[i];  // not very efficient query loading temporally, as assuming its not hotspot
    }

    __syncthreads();

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register int query_idx = 0;
        register float tmp_score = 0.;
        register bool no_more_load = false;

        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) {
                break;
            }
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
            register uint16_t *doc_segment = (uint16_t *)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                    // return;
                }
                while (query_idx < query_len && query_on_shm[query_idx] < doc_segment[j]) {
                    ++query_idx;
                }
                if (query_idx < query_len) {
                    tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
                }
            }
            __syncwarp();
        }
        scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    }
}

void doc_query_scoring_gpu(std::vector<std::vector<uint16_t>> &querys,
                           int start_doc_id,
                           std::vector<std::vector<uint16_t>> &docs,
                           std::vector<uint16_t> &lens,
                           std::vector<std::vector<int>> &indices,  // shape [querys.size(), TOPK]
                           std::vector<std::vector<float>> &scores  // shape [querys.size(), TOPK]
) {
    auto n_docs = docs.size();
    std::vector<float> s_scores(n_docs);
    std::vector<int> s_indices(n_docs);

    float *d_scores = nullptr;
    uint16_t *d_docs = nullptr, *d_query = nullptr;
    int *d_doc_lens = nullptr;

    // copy to device
    std::chrono::high_resolution_clock::time_point dat = std::chrono::high_resolution_clock::now();
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    hipMalloc(&d_scores, sizeof(float) * n_docs);
    hipMalloc(&d_doc_lens, sizeof(int) * n_docs);
    std::chrono::high_resolution_clock::time_point dat1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMalloc docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dat1 - dat).count() << " ms " << std::endl;

    // pre align docs -> h_docs [n_docs,MAX_DOC_SIZE], h_doc_lens_vec[n_docs]
    std::chrono::high_resolution_clock::time_point dgt = std::chrono::high_resolution_clock::now();
    uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
    memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    std::vector<int> h_doc_lens_vec(n_docs);
    for (int i = 0; i < docs.size(); i++) {
        for (int j = 0; j < docs[i].size(); j++) {
            auto group_sz = sizeof(group_t) / sizeof(uint16_t);
            auto layer_0_offset = j / group_sz;
            auto layer_0_stride = n_docs * group_sz;
            auto layer_1_offset = i;
            auto layer_1_stride = group_sz;
            auto layer_2_offset = j % group_sz;
            auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
            h_docs[final_offset] = docs[i][j];
        }
        h_doc_lens_vec[i] = docs[i].size();
    }
    std::chrono::high_resolution_clock::time_point dgt1 = std::chrono::high_resolution_clock::now();
    std::cout << "align group docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dgt1 - dgt).count() << " ms " << std::endl;

    std::chrono::high_resolution_clock::time_point dt = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    std::chrono::high_resolution_clock::time_point dt1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMemcpy H2D docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dt1 - dt).count() << " ms " << std::endl;

    std::chrono::high_resolution_clock::time_point dlt = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs, hipMemcpyHostToDevice);
    std::chrono::high_resolution_clock::time_point dlt1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMemcpy H2D doc_lens cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dlt1 - dlt).count() << " ms " << std::endl;

    // use one gpu device
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    hipSetDevice(0);

    for (auto &query : querys) {
        // init indices
        for (int i = 0; i < n_docs; ++i) {
            s_indices[i] = i + start_doc_id;
        }

        const size_t query_len = query.size();
        hipMalloc(&d_query, sizeof(uint16_t) * query_len);
        std::chrono::high_resolution_clock::time_point qt = std::chrono::high_resolution_clock::now();
        hipMemcpy(d_query, query.data(), sizeof(uint16_t) * query_len, hipMemcpyHostToDevice);
        std::chrono::high_resolution_clock::time_point qt1 = std::chrono::high_resolution_clock::now();
        std::cout << "hipMemcpy H2D query cost " << std::chrono::duration_cast<std::chrono::milliseconds>(qt1 - qt).count() << " ms " << std::endl;

        show_mem_usage();

        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;

        std::chrono::high_resolution_clock::time_point tt = std::chrono::high_resolution_clock::now();
        // hipLaunchKernel
        docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block>>>(d_docs,
                                                                          d_doc_lens, n_docs, d_query, query_len, d_scores);
        hipDeviceSynchronize();
        hipMemcpy(s_scores.data(), d_scores, sizeof(float) * n_docs, hipMemcpyDeviceToHost);
        std::chrono::high_resolution_clock::time_point tt1 = std::chrono::high_resolution_clock::now();
        std::cout << "docQueryScoringCoalescedMemoryAccessSampleKernel cost " << std::chrono::duration_cast<std::chrono::milliseconds>(tt1 - tt).count() << " ms " << std::endl;
        std::chrono::high_resolution_clock::time_point t = std::chrono::high_resolution_clock::now();
        int topk = s_scores.size() > TOPK ? TOPK : s_scores.size();
        // sort scores with Heap-based sort
        // todo: Bitonic sort by gpu
        std::partial_sort(s_indices.begin(), s_indices.begin() + topk, s_indices.end(),
                          [&s_scores, start_doc_id](const int &a, const int &b) {
                              if (s_scores[a - start_doc_id] != s_scores[b - start_doc_id]) {
                                  return s_scores[a - start_doc_id] > s_scores[b - start_doc_id];  // by score DESC
                              }
                              return a < b;  // the same score, by index ASC
                          });
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
        std::cout << "heap partial_sort cost " << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t).count() << " ms " << std::endl;

        std::vector<int> topk_doc_ids(s_indices.begin(), s_indices.begin() + topk);
        indices.emplace_back(topk_doc_ids);

        std::vector<float> topk_scores(topk_doc_ids.size());
        int i = 0;
        for (auto doc_id : topk_doc_ids) {
            topk_scores[i++] = s_scores[doc_id - start_doc_id];
        }
        scores.emplace_back(topk_scores);

        hipFree(d_query);
    }

    hipFree(d_docs);
    hipFree(d_scores);
    hipFree(d_doc_lens);
    free(h_docs);
}
