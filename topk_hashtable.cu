#include "hip/hip_runtime.h"
#include "hashtable.h"
#include "helper.h"
#include "topk.h"

// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#built-in-vector-types
typedef uint4 group_t;  // cuda uint4: 4 * uint (sizeof(uint4)=16 128bit)

// intersection(query,doc): query[i] == doc[j](0 <= i < query_size, 0 <= j < doc_size)
// score = total_intersection(query,doc) / max(query_size, doc_size)
void __global__ docQueryScoringCoalescedMemoryHashTableKernel(
    const __restrict__ uint16_t *docs,
    const int *doc_lens, const size_t n_docs,
    GPUHashTable<uint16_t, unsigned int>::device_view const d_query, const int query_len, float *scores) {
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;

    if (tid >= n_docs) {
        return;
    }

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register float tmp_score = 0.;
        register bool no_more_load = false;

        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) {
                break;
            }
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id];  // tid
            register uint16_t *doc_segment = (uint16_t *)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                }
                auto q_item_cn = d_query.lookup(doc_segment[j]);
                if (q_item_cn > 0) {
                    tmp_score += float(q_item_cn);
                }
            }
            __syncwarp();
        }
        scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]);  // tid
    }
}

void doc_query_scoring_gpu(std::vector<std::vector<uint16_t>> &querys,
                           int start_doc_id,
                           std::vector<std::vector<uint16_t>> &docs,
                           std::vector<uint16_t> &lens,
                           std::vector<std::vector<int>> &indices,  // shape [querys.size(), TOPK]
                           std::vector<std::vector<float>> &scores  // shape [querys.size(), TOPK]
) {
    auto n_docs = docs.size();
    std::vector<float> s_scores(n_docs);
    std::vector<int> s_indices(n_docs);

    float *d_scores = nullptr;
    uint16_t *d_docs = nullptr, *d_query = nullptr;
    int *d_doc_lens = nullptr;

    // copy to device
    std::chrono::high_resolution_clock::time_point dat = std::chrono::high_resolution_clock::now();
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    hipMalloc(&d_scores, sizeof(float) * n_docs);
    hipMalloc(&d_doc_lens, sizeof(int) * n_docs);
    std::chrono::high_resolution_clock::time_point dat1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMalloc docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dat1 - dat).count() << " ms " << std::endl;

    // pre align docs -> h_docs [n_docs,MAX_DOC_SIZE], h_doc_lens_vec[n_docs]
    std::chrono::high_resolution_clock::time_point dgt = std::chrono::high_resolution_clock::now();
    uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
    memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    std::vector<int> h_doc_lens_vec(n_docs);
    for (int i = 0; i < docs.size(); i++) {
        for (int j = 0; j < docs[i].size(); j++) {
            auto group_sz = sizeof(group_t) / sizeof(uint16_t);
            auto layer_0_offset = j / group_sz;
            auto layer_0_stride = n_docs * group_sz;
            auto layer_1_offset = i;
            auto layer_1_stride = group_sz;
            auto layer_2_offset = j % group_sz;
            auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
            h_docs[final_offset] = docs[i][j];
        }
        h_doc_lens_vec[i] = docs[i].size();
    }
    std::chrono::high_resolution_clock::time_point dgt1 = std::chrono::high_resolution_clock::now();
    std::cout << "align group docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dgt1 - dgt).count() << " ms " << std::endl;

    std::chrono::high_resolution_clock::time_point dt = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    std::chrono::high_resolution_clock::time_point dt1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMemcpy H2D docs cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dt1 - dt).count() << " ms " << std::endl;

    std::chrono::high_resolution_clock::time_point dlt = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs, hipMemcpyHostToDevice);
    std::chrono::high_resolution_clock::time_point dlt1 = std::chrono::high_resolution_clock::now();
    std::cout << "hipMemcpy H2D doc_lens cost " << std::chrono::duration_cast<std::chrono::milliseconds>(dlt1 - dlt).count() << " ms " << std::endl;

    // use one gpu device
    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);
    hipSetDevice(0);

    for (auto &query : querys) {
        // init indices
        for (int i = 0; i < n_docs; ++i) {
            s_indices[i] = i + start_doc_id;
        }

        const size_t query_len = query.size();
        hipMalloc(&d_query, sizeof(uint16_t) * query_len);
        std::chrono::high_resolution_clock::time_point qt = std::chrono::high_resolution_clock::now();
        hipMemcpy(d_query, query.data(), sizeof(uint16_t) * query_len, hipMemcpyHostToDevice);
        std::chrono::high_resolution_clock::time_point qt1 = std::chrono::high_resolution_clock::now();
        std::cout << "hipMemcpy H2D query cost " << std::chrono::duration_cast<std::chrono::milliseconds>(qt1 - qt).count() << " ms " << std::endl;

        // build query hash table
        GPUHashTable<uint16_t, unsigned int> q_ht(MAX_DOC_SIZE);
        q_ht.insert_unique_cn_many(d_query, query_len);

        show_mem_usage();

        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;

        std::chrono::high_resolution_clock::time_point tt = std::chrono::high_resolution_clock::now();
        // hipLaunchKernel
        auto d_ht_view = q_ht.get_device_view();
        docQueryScoringCoalescedMemoryHashTableKernel<<<grid, block>>>(d_docs,
                                                                       d_doc_lens, n_docs, d_ht_view, query_len, d_scores);
        hipDeviceSynchronize();
        hipMemcpy(s_scores.data(), d_scores, sizeof(float) * n_docs, hipMemcpyDeviceToHost);
        std::chrono::high_resolution_clock::time_point tt1 = std::chrono::high_resolution_clock::now();
        std::cout << "docQueryScoringCoalescedMemoryAccessSampleKernel cost " << std::chrono::duration_cast<std::chrono::milliseconds>(tt1 - tt).count() << " ms " << std::endl;
        std::chrono::high_resolution_clock::time_point t = std::chrono::high_resolution_clock::now();
        int topk = s_scores.size() > TOPK ? TOPK : s_scores.size();
        // sort scores with Heap-based sort
        // todo: Bitonic sort by gpu
        std::partial_sort(s_indices.begin(), s_indices.begin() + topk, s_indices.end(),
                          [&s_scores, start_doc_id](const int &a, const int &b) {
                              if (s_scores[a - start_doc_id] != s_scores[b - start_doc_id]) {
                                  return s_scores[a - start_doc_id] > s_scores[b - start_doc_id];  // by score DESC
                              }
                              return a < b;  // the same score, by index ASC
                          });
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
        std::cout << "heap partial_sort cost " << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t).count() << " ms " << std::endl;

        std::vector<int> topk_doc_ids(s_indices.begin(), s_indices.begin() + topk);
        indices.emplace_back(topk_doc_ids);

        std::vector<float> topk_scores(topk_doc_ids.size());
        int i = 0;
        for (auto doc_id : topk_doc_ids) {
            topk_scores[i++] = s_scores[doc_id - start_doc_id];
        }
        scores.emplace_back(topk_scores);

        hipFree(d_query);
    }

    hipFree(d_docs);
    hipFree(d_scores);
    hipFree(d_doc_lens);
    delete[] h_docs;
}
