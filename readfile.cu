#include "hip/hip_runtime.h"
#include "helper.h"
#include "readfile.h"
#include "topk.h"

__global__ void docsKernel(cudf::column_device_view const d_docs, const size_t n_docs, uint16_t *out_docs, uint16_t *out_doc_len) {
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;

#ifdef DEBUG
    printf("tid:%d tnum:%d GPU from block(%d, %d, %d), thread(%d, %d, %d)\n ",
           tid, tnum,
           blockIdx.x,
           blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z);
#endif
    if (tid >= n_docs) {
        return;
    }
    auto docs = cudf::detail::lists_column_device_view(d_docs);
    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        auto offset_s = docs.offset_at(doc_id);
        auto offset_e = docs.offset_at(doc_id + 1);
        auto sub_view = docs.child().slice(offset_s, offset_e - offset_s);
        // printf("\ntid:%d docid:%d s:%d e:%d sub_view_size:%d\n", tid, doc_id, offset_s, offset_e, sub_view.size());
        // if (doc_id == 2){
        for (auto i = 0; i < sub_view.size(); i++) {
            auto const item = sub_view.element<cudf::string_view>(i);
            int num = h_atoi(item.data());
            // printf("%d,", num);
            out_docs[doc_id * MAX_DOC_SIZE + i] = num;
        }
        //}
        out_doc_len[doc_id] = sub_view.size();
    }
}

void load_file_cudf_chunk(std::string docs_file_name, std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &doc_lens) {
    unsigned int buffsize = CHUNK_SIZE;
    int count = 0;
    int readcnt = 0;
    unsigned int doccnt = 0;
    char *buff = new char[buffsize];

    FILE *fd = fopen(docs_file_name.c_str(), "rb");
    // fseek(fd, 0, SEEK_END);
    // std::cout << "file size: " << ftell(fd) << std::endl;
    // fseek(fd, 0, SEEK_SET);
    std::cout << "chunk size: " << buffsize << std::endl;

    while (!feof(fd)) {
        memset(buff, 0, buffsize);
        count = fread(buff, sizeof(char), buffsize, fd);
        auto cur_pos = ftell(fd);
        std::string chunk_buff(buff);
        auto offset = chunk_buff.find_last_of("\n");
        if (!feof(fd) && offset != std::string::npos) {
            chunk_buff.erase(offset + 1);
            fseek(fd, cur_pos - (buffsize - offset) + 1, SEEK_SET);
        }
        std::cout << " fread size: " << count << std::endl;
        // std::cout << " buffer: " << chunk_buff << std::endl;

        hipStream_t stream;
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        rmm::cuda_stream_view stream_view(stream);
        // cudf multibyte_split
        auto delimiter = "\n";
        cudf::io::text::parse_options options;
        options.strip_delimiters = false;
        auto source = cudf::io::text::make_source(chunk_buff);
        auto lines = cudf::io::text::multibyte_split(*source, delimiter, options, stream_view);
        auto vec_lines = cudf::strings::split_record(lines->view(), cudf::string_scalar(","), -1, stream_view);
        auto const d_col = cudf::column_device_view::create(vec_lines->view());

        auto n_docs = lines->size();
        uint16_t *d_docs = nullptr;
        uint16_t *d_doc_lens = nullptr;
        hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
        hipMemset(d_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
        hipMalloc(&d_doc_lens, sizeof(uint16_t) * n_docs);
        // hipMemset(d_doc_lens, 0, sizeof(uint16_t) * n_docs);

        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;
        docsKernel<<<grid, block, 0, stream_view.value()>>>(*d_col, n_docs, d_docs, d_doc_lens);

        uint16_t *h_docs = nullptr;
        uint16_t *h_doc_lens = nullptr;
#ifdef PINNED_MEMORY
        // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1gb65da58f444e7230d3322b6126bb4902
        hipHostMalloc(&h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);  // hipHostMallocDefault
        // hipHostAlloc(&h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipHostMallocDefault);
        hipHostAlloc(&h_doc_lens, sizeof(int) * n_docs, hipHostMallocDefault);
        // hipHostAlloc(&h_doc_offsets_vec, sizeof(int) * n_docs, hipHostMallocDefault);
#else
        h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
        h_doc_lens = new uint16_t[n_docs];
#endif
        // memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
        // memset(h_doc_lens, 0, sizeof(uint16_t) * n_docs);
        hipMemcpyAsync(h_docs, d_docs, sizeof(uint16_t) * n_docs * MAX_DOC_SIZE, hipMemcpyDeviceToHost, stream_view.value());
        hipMemcpyAsync(h_doc_lens, d_doc_lens, sizeof(uint16_t) * n_docs, hipMemcpyDeviceToHost, stream_view.value());
        hipStreamSynchronize(stream_view.value());

#ifdef DEBUG
        std::cout << "h_docs:" << std::endl;
        print2d_uint16(h_docs, n_docs, MAX_DOC_SIZE);
        std::cout << "h_doc_lens:" << std::endl;
        print1d_uint16(h_doc_lens, n_docs);
#endif

        for (int i = 0; i < n_docs; i++) {
            std::vector<uint16_t> vec_docs;
            vec_docs.reserve(h_doc_lens[i]);
            vec_docs.insert(vec_docs.end(), h_docs + i * MAX_DOC_SIZE, h_docs + i * MAX_DOC_SIZE + h_doc_lens[i]);
            docs.emplace_back(vec_docs);
        }
        doc_lens.insert(doc_lens.end(), h_doc_lens, h_doc_lens + n_docs);

        hipStreamDestroy(stream_view.value());
        hipFree(d_docs);
        hipFree(d_doc_lens);
#ifdef PINNED_MEMORY
        hipHostFree(h_docs);
        hipHostFree(h_doc_lens);
#else
        delete[] h_docs;
        delete[] h_doc_lens;
#endif

        doccnt += n_docs;
        readcnt++;
    }
    std::cout << "readcnt: " << readcnt << std::endl;
    std::cout << "doccnt: " << doccnt << std::endl;
#ifdef DEBUG
    std::cout << "docs:" << std::endl;
    for (auto doc : docs) {
        print(doc);
    }
    std::cout << "doc_lens:" << std::endl;
    print(doc_lens);
#endif

    free(buff);
    fclose(fd);
}

void load_file_stream_cudf_chunk(std::string docs_file_name, std::vector<std::vector<uint16_t>> &docs, std::vector<uint16_t> &doc_lens) {
    unsigned int buffsize = CHUNK_SIZE;
    int count = 0;
    int readcnt = 0;
    unsigned int doccnt = 0;
    char *buff = new char[buffsize];
    FILE *fd = fopen(docs_file_name.c_str(), "rb");
    fseek(fd, 0, SEEK_END);
    auto file_size = ftell(fd);
    std::cout << "file size: " << file_size << std::endl;
    std::cout << "chunk size: " << buffsize << std::endl;
    fseek(fd, 0, SEEK_SET);

    auto split_cn = (file_size + buffsize - 1) / buffsize;
    hipStream_t *doc_streams = new hipStream_t[split_cn];
    std::cout << " Creating " << split_cn << " CUDA streams." << std::endl;
    for (int i = 0; i < split_cn; i++) {
        CUDA_CALL(hipStreamCreate(&doc_streams[i]));
    }

    while (!feof(fd)) {
        memset(buff, 0, buffsize);
        count = fread(buff, sizeof(char), buffsize, fd);
        auto cur_pos = ftell(fd);
        std::string chunk_buff(buff);
        auto offset = chunk_buff.find_last_of("\n");
        if (!feof(fd) && offset != std::string::npos) {
            chunk_buff.erase(offset + 1);
            fseek(fd, cur_pos - (buffsize - offset) + 1, SEEK_SET);
        }
        std::cout << " fread size: " << count << std::endl;
        // std::cout << " buffer: " << chunk_buff << std::endl;

        // cudf multibyte_split
        auto delimiter = "\n";
        cudf::io::text::parse_options options;
        options.strip_delimiters = false;
        auto source = cudf::io::text::make_source(chunk_buff);
        // todo:: need multibyte_split support stream
        auto lines = cudf::io::text::multibyte_split(*source, delimiter, options, cudf::get_default_stream());
        auto vec_lines = cudf::strings::split_record(lines->view(), cudf::string_scalar(","));
        auto const d_col = cudf::column_device_view::create(vec_lines->view());
        // todo: launch docsKernel with stream

        doccnt += lines->size();
        readcnt++;
    }
    std::cout << "readcnt: " << readcnt << std::endl;
    std::cout << "doccnt: " << doccnt << std::endl;

    free(doc_streams);
    free(buff);
    fclose(fd);
}

#ifdef PIO_TOPK
typedef std::tuple<std::vector<std::vector<int>>, std::vector<std::vector<float>>> tupleIdScores;

void load_file_cudf_chunk_topk(const std::string docs_file_name,
                               std::vector<std::vector<uint16_t>> &queries,
                               std::vector<std::vector<int>> &indices,
                               std::vector<std::vector<float>> &scores) {
    std::vector<std::vector<int>> q_indices(queries.size());
    std::vector<std::vector<float>> q_scores(queries.size());

    unsigned int buffsize = CHUNK_SIZE;
    int count = 0;
    int readcnt = 0;
    unsigned int doccnt = 0;
    char *buff = new char[buffsize];

    FILE *fd = fopen(docs_file_name.c_str(), "rb");
    // fseek(fd, 0, SEEK_END);
    // std::cout << "file size: " << ftell(fd) << std::endl;
    // fseek(fd, 0, SEEK_SET);
    std::cout << "chunk size: " << buffsize << std::endl;

#ifdef PIO_CPU_CONCURRENCY
    int concurrency = std::thread::hardware_concurrency();
    std::cout << "hardware concurrency:" << concurrency << std::endl;
    ThreadPool pool(concurrency);
    std::vector<std::future<tupleIdScores>> results;
#endif

    while (!feof(fd)) {
        memset(buff, 0, buffsize);
        count = fread(buff, sizeof(char), buffsize, fd);
        auto cur_pos = ftell(fd);
        std::string chunk_buff(buff);
        auto offset = chunk_buff.find_last_of("\n");
        if (!feof(fd) && offset != std::string::npos) {
            chunk_buff.erase(offset + 1);
            fseek(fd, cur_pos - (buffsize - offset) + 1, SEEK_SET);
        }
        std::cout << " fread size: " << count << std::endl;
        // std::cout << " buffer: " << chunk_buff << std::endl;

        hipStream_t stream;
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
        rmm::cuda_stream_view stream_view(stream);
        // cudf multibyte_split
        auto delimiter = "\n";
        cudf::io::text::parse_options options;
        options.strip_delimiters = false;
        auto source = cudf::io::text::make_source(chunk_buff);
        auto lines = cudf::io::text::multibyte_split(*source, delimiter, options, stream_view);
        auto vec_lines = cudf::strings::split_record(lines->view(), cudf::string_scalar(","), -1, stream_view);
        auto d_col = cudf::column_device_view::create(vec_lines->view());
        auto n_docs = lines->size();

#ifdef PIO_CPU_CONCURRENCY
        // std::unique_ptr https://github.com/progschj/ThreadPool/issues/93 lambda maybe always upgrade
        auto f = [&queries, doccnt, n_docs, col = std::move(d_col), &stream_view]() mutable {
            std::vector<std::vector<int>> sub_topk_indices;
            std::vector<std::vector<float>> sub_topk_scores;
            doc_query_scoring_gpu(queries, doccnt, n_docs, std::move(*col), sub_topk_indices, sub_topk_scores, stream_view);
            hipStreamDestroy(stream_view.value());
            return std::make_tuple(sub_topk_indices, sub_topk_scores);
        };
        results.emplace_back(pool.enqueue(std::move(f)));
#else
        std::vector<std::vector<int>> sub_topk_indices;
        std::vector<std::vector<float>> sub_topk_scores;
        doc_query_scoring_gpu(queries, doccnt, n_docs, *d_col, sub_topk_indices, sub_topk_scores, stream_view);
        for (auto i = 0; i < queries.size(); i++) {
            q_indices[i].insert(q_indices[i].end(), sub_topk_indices[i].begin(), sub_topk_indices[i].end());
            q_scores[i].insert(q_scores[i].end(), sub_topk_scores[i].begin(), sub_topk_scores[i].end());
        }
        hipStreamDestroy(stream_view.value());
#endif

        doccnt += n_docs;
        readcnt++;
    }
    std::cout << "readcnt: " << readcnt << std::endl;
    std::cout << "doccnt: " << doccnt << std::endl;

#ifdef PIO_CPU_CONCURRENCY
    for (auto &&result : results) {
        auto res = result.get();
        for (auto i = 0; i < queries.size(); i++) {
            q_indices[i].insert(q_indices[i].end(), std::get<0>(res)[i].begin(), std::get<0>(res)[i].end());
            q_scores[i].insert(q_scores[i].end(), std::get<1>(res)[i].begin(), std::get<1>(res)[i].end());
        }
    }
#endif

    // sort topk
    for (auto i = 0; i < queries.size(); i++) {
        std::unordered_map<int, int> indices_map;
        for (auto j = 0; j < q_indices[i].size(); j++) {
            indices_map[q_indices[i][j]] = j;
        }
        int topk = q_indices[i].size() > TOPK ? TOPK : q_indices[i].size();
        std::partial_sort(q_indices[i].begin(), q_indices[i].begin() + topk, q_indices[i].end(),
                          [&q_scores, i, &indices_map](const int &a, const int &b) {
                              if (q_scores[i][indices_map[a]] != q_scores[i][indices_map[b]]) {
                                  return q_scores[i][indices_map[a]] > q_scores[i][indices_map[b]];  // by score DESC
                              }
                              return a < b;  // the same score, by index ASC
                          });

        std::vector<int> topk_doc_ids(q_indices[i].begin(), q_indices[i].begin() + topk);
        indices.emplace_back(topk_doc_ids);

        std::vector<float> topk_scores(topk_doc_ids.size());
        int id = 0;
        for (auto doc_id : topk_doc_ids) {
            topk_scores[id++] = q_scores[i][indices_map[doc_id]];
        }
        scores.emplace_back(topk_scores);
    }

    free(buff);
    fclose(fd);
}
#endif